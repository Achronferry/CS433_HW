#include "hip/hip_runtime.h"
/*
This is the cuda version of 2d-convolution with multiple thread blocks.
In this problem, our output with size of [N, F, H_, W_];
So we divides the task into (H_ * W_) blocks, and each nlock has (N * F) threads.
*/
#include "matr_def.h"


__global__ void conv2d_cuda_optim_kernel(float *out_matr, float *fm_matr, float *kn_matr,
                                    int in_channel, int out_channel, int height, int width, 
                                    int ksize_x, int ksize_y);

void Conv2D_cuda_optim(Matrix &out, Matrix fm, Matrix kn) {
    fm.cuda(); kn.cuda(); out.cuda();
    dim3 block_sz(out.d1, out.d2);
    dim3 grid_sz(out.d3, out.d4);
    conv2d_cuda_optim_kernel<<<grid_sz,block_sz>>>(out.element, fm.element, kn.element, 
                                            kn.d2, kn.d1, fm.d3, fm.d4, kn.d3, kn.d4);
    out.cpu();
}


__global__ void conv2d_cuda_optim_kernel(float *out_matr, float *fm_matr, float *kn_matr,
                                    int in_channel, int out_channel, int height, int width, 
                                    int ksize_x, int ksize_y) {
    int batch_id = threadIdx.x, channel_id = threadIdx.y;
    int row = blockIdx.x, col = blockIdx.y;
    float cell_value = 0;
    for (int c = 0; c < in_channel; c++) // each in-channel
        for (int i = 0; i < ksize_x; i++) 
            for (int j = 0; j < ksize_y; j++) // each lacation of a kernel 
            cell_value += kn_matr[channel_id*in_channel*ksize_x*ksize_y + c*ksize_x*ksize_y + i*ksize_y + j] * 
                    fm_matr[batch_id*in_channel*height*width + c*height*width + (row+i)*width + (col+j)];
    // printf("[%d,%d,%d,%d] = %f\n", batch_id, channel_id, row, col, cell_value);
    out_matr[batch_id*out_channel*(height - ksize_x + 1)*(width - ksize_y + 1) + 
                channel_id*(height - ksize_x + 1)*(width - ksize_y + 1) +
                row*(width - ksize_y + 1) + col] = cell_value;
}

int main() {
    //Initialize Matrix
    Matrix fm(N, C, H, W), kn(F, C, K, K);
    Matrix out(N, F, H-K+1, W-K+1);
    Matrix truth(N, F, H-K+1, W-K+1);
    fm.fill_value(1.0);
    kn.fill_value(0.5);
    truth.fill_value(288.0);
    printf("The feature map is filled with %f;\n",*fm.get(1,2,3,4));
    printf("The kernel is filled with %f;\n",*kn.get(1,2,3,4));
    clock_t st,ed;
    st = clock();
    Conv2D_cuda_optim(out, fm, kn);
    ed = clock();
    printf("It takes %f ms to calculate the convolution...", (double)(ed-st)/CLOCKS_PER_SEC * 1000);
    if (out == truth)
        printf("Result is correct! (%f)\n", *out.get(1,2,3,4));
    else
        printf("Result is wrong! (%f)\n", *out.get(1,2,3,4));
}
